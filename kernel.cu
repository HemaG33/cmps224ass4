#include "hip/hip_runtime.h"
#include "common.h"

#include "timer.h"

#define IN_TILE_DIM 32
#define OUT_TILE_DIM ((IN_TILE_DIM) - 2*(FILTER_RADIUS))

__constant__ float filter_c[FILTER_DIM][FILTER_DIM];

__global__ void convolution_tiled_kernel(float* input, float* output, unsigned int width, unsigned int height) {

    __shared__ float inputTile[IN_TILE_DIM + FILTER_DIM - 1][IN_TILE_DIM + FILTER_DIM - 1];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row_i = row - FILTER_RADIUS;
    int col_i = col - FILTER_RADIUS;

    // Load input tile to shared memory
    if (row_i >= 0 && row_i < height && col_i >= 0 && col_i < width) {
        inputTile[threadIdx.y][threadIdx.x] = input[row_i * width + col_i];
    } else {
        inputTile[threadIdx.y][threadIdx.x] = 0.0f;  // Boundary condition: pad with zeros
    }

    __syncthreads(); 

    float sum = 0.0f;
    for (int i = 0; i < FILTER_DIM; ++i) {
        for (int j = 0; j < FILTER_DIM; ++j) {
            sum += filter_c[i][j] * inputTile[threadIdx.y + i][threadIdx.x + j];
        }
    }

    // Store the result to output if within valid range
    if (row < height && col < width) {
        output[row * width + col] = sum;
    }
}

void copyFilterToGPU(float filter[][FILTER_DIM]) {
    // Copy filter to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(filter_c), filter, FILTER_DIM * FILTER_DIM * sizeof(float));
}

void convolution_tiled_gpu(float* input_d, float* output_d, unsigned int width, unsigned int height) {
    // Call kernel
    dim3 numThreadsPerBlock(IN_TILE_DIM, IN_TILE_DIM);
    dim3 numBlocks((width + IN_TILE_DIM - 1) / IN_TILE_DIM, (height + IN_TILE_DIM - 1) / IN_TILE_DIM);
    convolution_tiled_kernel<<<numBlocks, numThreadsPerBlock>>>(input_d, output_d, width, height);
}

